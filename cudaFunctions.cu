#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "myProto.h"

__global__  void preform_task(mutant * mutants, char * seq1, char * seq2, double * W, int min_max, int start, int end, int maxOffset, char * temp_seq2);
__device__ mutant check_cuda(int offset, char * seq1, char * seq2, double * W, int min_max, int start, int end, int seq2_length, char * temp_seq2);
__device__ double alignment_score_cuda(char * seq1, char * seq2, double * W, char c, int index);
__device__ int check_group_cuda(char c1, char c2);
__device__ int check_conservative_cuda(char c1, char c2);
__device__ int check_semi_conservative_cuda(char c1, char c2);
__device__ int my_strchr(const char *str, char ch);
__device__ char * my_strcpy(char *dest, const char *src);

__global__  void preform_task(mutant * mutants, char * seq1, char * seq2, double * W, int min_max, int start, int end, int maxOffset, int seq2_length, char * temp_seq2) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Increment the proper value of the arrray according to thread ID 
    if (i < maxOffset){
        mutants[i] = check_cuda(i, seq1, seq2, W, min_max, start, end, seq2_length, temp_seq2);
    }
}

int computeOnGPU(mutant * mutants, char * seq1, char * seq2, double * W, int min_max, int start, int end, int maxOffset, int seq1_length, int seq2_length){
    char temp_str [SEQ2_SIZE];
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    hipError_t err1 = hipSuccess;
    hipError_t err2 = hipSuccess;
    hipError_t err3 = hipSuccess;
    hipError_t err4 = hipSuccess;
    hipError_t err5 = hipSuccess;

    size_t size_seq1 = (seq1_length + 1) * sizeof(char);
    size_t size_seq2 = (seq2_length + 1) * sizeof(char);
    size_t size_W = 4*sizeof(double);
    size_t size_mutants = maxOffset * sizeof(mutant);

    // Allocate memory on GPU to copy the data from the host
    char *d_seq1;
    char *d_seq2;
    char *d_temp_seq2;
    double * d_W;
    mutant *d_mutants;
    err1 = hipMalloc((void **)&d_seq1, size_seq1);
    err2 = hipMalloc((void **)&d_seq2, size_seq2);
    err3 = hipMalloc((void **)&d_mutants, size_mutants);
    err4 = hipMalloc((void **)&d_W, size_W);
    err5 = hipMalloc((void **)&d_temp_seq2, SEQ2_SIZE * sizeof(char));
   

    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess ||err5 != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err1 = hipMemcpy(d_seq1, seq1, size_seq1, hipMemcpyHostToDevice);
    err2 = hipMemcpy(d_seq2, seq2, size_seq2, hipMemcpyHostToDevice);
    err3 = hipMemcpy(d_mutants, mutants, size_mutants, hipMemcpyHostToDevice);
    err4 = hipMemcpy(d_W, W, size_W, hipMemcpyHostToDevice);
    err5 = hipMemcpy(d_temp_seq2, temp_str, SEQ2_SIZE * sizeof(char), hipMemcpyHostToDevice);

    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(maxOffset + threadsPerBlock - 1) / threadsPerBlock;
	
    preform_task<<<blocksPerGrid, threadsPerBlock>>>(d_mutants, d_seq1, d_seq2, d_W, min_max, start, end, maxOffset, seq2_length, d_temp_seq2);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(mutants, d_mutants, size_mutants, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_mutants) != hipSuccess || hipFree(d_seq1) != hipSuccess || hipFree(d_W) != hipSuccess || hipFree(d_seq2) != hipSuccess || hipFree(d_temp_seq2) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    //create mutant sequence	
	strcpy(temp_str, seq2);
    for(int i = 0 ; i < maxOffset ; i++){
        temp_str[mutants[i].pos] = mutants[i].c;
        strcpy(mutants[i].seq, temp_str);
	strcpy(temp_str, seq2);
    }

    return 0;
}

//perform the required task
__device__ mutant check_cuda(int offset, char * seq1, char * seq2, double * W, int min_max, int start, int end, int seq2_length, char * temp_seq2){
   my_strcpy(temp_seq2, seq2);
   mutant m;
   double max_score = -100000000;
   double min_score = 100000000;
   double current_score = 0;
   char * p1 = seq1 + offset;//adding the current offset to seq1 
   int index = start;//the required start index for seq2
   int group;
   char original_c;
   char c;
   while(seq2[index] != '\0' && index <= end){//continue running the loop until rech end of seq2 or until reaching the required end index for seq2
       c = 'A';
       original_c = seq2[index];//save the the original char from seq2 for reconstruction
       for (int i = 0 ; i < 26 ; i++){//run through all letters for mutant check
           group = check_group_cuda(original_c, c);//check if the current letter (c) can be a mutant according to the Substitution Rules (space or point)
           if(group == POINT ||group == SPACE || group == STAR){
               current_score = alignment_score_cuda(p1, temp_seq2, W, c, index);//calculate the alignment score
               if (min_max){//if searching for maximum alignment score value
                   if(current_score > max_score){
                       max_score = current_score;
                       m.score = max_score;
                       m.offset = offset;
		       m.pos = index;
                       m.c = c;
                   }
               }
               else if (current_score < min_score){//if searching for minimum alignment score value
                       min_score = current_score;
                       m.score = min_score;
                       m.offset = offset;
		       m.pos = index;
                       m.c = c;
               }
           }
           c++;
           
       }
       index++;
   }

   return m;
}

__device__ double alignment_score_cuda(char * seq1, char * seq2, double * W, char c, int index){//calculating Sequences alignment score
    int numStars = 0, numcolons = 0, numPoints = 0, numSpace = 0;
    char * p1 = seq1;
    char * p2 = seq2;
    int pos = 0 ;
    while(*p2 != '\0' && *p1 != '\0'){
	if (pos == index){
		switch (check_group_cuda(*p1, c))//check the sign of each pair
		{
		case STAR:
		    numStars++;
		    break;
		case COLON:
		    numcolons++;
		    break;
		case POINT:
		    numPoints++;
		    break;
		case SPACE:
		    numSpace++;
		    break;
		default:
		    break;
		}
	}
	else{
	        switch (check_group_cuda(*p1, *p2))//check the sign of each pair
		{
		case STAR:
		    numStars++;
		    break;
		case COLON:
		    numcolons++;
		    break;
		case POINT:
		    numPoints++;
		    break;
		case SPACE:
		    numSpace++;
		    break;
		default:
		    break;
		}
	}
        p1++;
        p2++;
	pos++;
    }
    return W[0]*numStars - W[1]*numcolons - W[2]*numPoints - W[3]*numSpace;//formula for the Alignment Score
}

__device__ int check_group_cuda(char c1, char c2){//check the sign of a given pair of latters
    if (c1 == c2)
        return STAR;
    if(check_conservative_cuda(c1, c2))
        return COLON;
    else if (check_semi_conservative_cuda(c1, c2))
        return POINT;
    else
        return SPACE;
}

__device__ int check_conservative_cuda(char c1, char c2){//check if a pair of latters are in the same conservative group
    const char * conservative[9] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
    for (int i = 0; i < 9 ; i++)
        if(my_strchr(conservative[i], c1) != NULL && my_strchr(conservative[i], c2)!= NULL)
            return 1;
    return 0;
}

__device__ int check_semi_conservative_cuda(char c1, char c2){//check if a pair of latters are in the same semi conservative group
    const char * semi_conservative[11] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
    for (int i = 0; i < 11 ; i++)
        if(my_strchr(semi_conservative[i], c1) != NULL  && my_strchr(semi_conservative[i], c2) != NULL)
            return 1;
    return 0;
}

__device__ int my_strchr(const char *str, char ch)//function to preform strchr on cuda
{
for (;; str++) {
        if (*str == ch) return 1;
        if (!*str) return 0;
        }
}

__device__ char * my_strcpy(char *dest, const char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != 0);
  return dest;
}
